#include "hip/hip_runtime.h"
#include "atomicAddTemplate.cu"
#include "mathematicalFunctions.cu"

#ifndef HELPERS
#define HELPERS
template<typename T>
__device__ inline T computeDist(const T* vec1, const T* vec2) {
   return MathematicalFunc::cudaSqrt((vec2[0] - vec1[0]) * (vec2[0] - vec1[0])
                   + (vec2[1] - vec1[1]) * (vec2[1] - vec1[1])
                   + (vec2[2] - vec1[2]) * (vec2[2] - vec1[2]));
}

template<typename T>
__device__ inline T dotProduct3(const T* vec1, const T* vec2) {
   return vec1[0]*vec2[0] + vec1[1]*vec2[1] + vec1[2]*vec2[2];
}

template<typename T>
__device__ inline T dotProduct3Static(const T vec1[], const T* vec2) {
   return vec1[0]*vec2[0] + vec1[1]*vec2[1] + vec1[2]*vec2[2];
}

template<typename T>
__device__ inline void vectorMinus3(T result[], const T* vec1, const T vec2[]) {
   result[0] = vec1[0] - vec2[0];
   result[1] = vec1[1] - vec2[1];
   result[2] = vec1[2] - vec2[2];
}

template<typename T>
__device__ inline T hypot(const T val1, const T val2) {
   return MathematicalFunc::cudaSqrt(val1*val1 + val2*val2);
}

template<typename T>
__device__ inline T norm2(const T vec[]) {
   return MathematicalFunc::cudaSqrt(vec[0]*vec[0] + vec[1]*vec[1] + vec[2]*vec[2]);
}

template<typename T>
__device__ inline void positiveXY(const T exponent, const T weight,
                                  T &resultReal, T &resultImag) {                           
   resultReal += weight*MathematicalFunc::cudaCos(exponent);
   resultImag += weight*MathematicalFunc::cudaSin(exponent);
}

template<typename T>
__device__ inline void negativeXY(const T exponent, const T weight,
                                  T &resultReal, T &resultImag) {
   resultReal -= weight*MathematicalFunc::cudaCos(exponent);
   resultImag -= weight*MathematicalFunc::cudaSin(exponent);
}

template<typename T>
__device__ inline void positiveXYDivided(const T exponent, const T weight,
        T &resultReal, T &resultImag, const T dist, const T wavelength) {
  if(dist == 0) {
    resultImag -= weight * wavelength;
  } else {
    resultReal += weight*(MathematicalFunc::cudaCos(exponent) - 1)/dist;
    resultImag += weight*MathematicalFunc::cudaSin(exponent)/dist;
  }
}

template<typename T>
__device__ inline void negativeXYDivided(const T exponent, const T weight,
        T &resultReal, T &resultImag, const T dist, const T wavelength) {
  if(dist == 0) {
    resultImag += weight * wavelength;
  } else {
    resultReal -= weight*(MathematicalFunc::cudaCos(exponent) - 1)/dist;
    resultImag -= weight*MathematicalFunc::cudaSin(exponent)/dist;
  }
}

template<typename T>
__device__ inline T RWGl10(const T sn, const T sp, const T Rn, const T Rp) {
  T sgn = sp < 0 ? -1 : 1;
  return sgn*MathematicalFunc::cudaLog((sp + sgn*Rp) / (sn + sgn*Rn)); 
}

template<typename T>
__device__ inline T RWG111(const T sn, const T sp, const T R0, const T Rn, const T Rp, const T I) {
   return 0.5 * (sp*Rp - sn*Rn + R0*R0*I);
}

template<typename T>
__device__ inline T RWGs00(const T sn, const T sp, const T t0, const T w0, const T R0, const T Rn, const T Rp) {
   if(MathematicalFunc::cudaAbs(t0) < 1e-16) {
      return 0;
   } else {
      T absW0 = MathematicalFunc::cudaAbs(w0);
      T R02 = R0*R0;
      return MathematicalFunc::cudaAtan(t0*sp / (R02 + absW0*Rp)) - MathematicalFunc::cudaAtan(t0*sn / (R02 + absW0*Rn));
   }
}

// template<typename T>
// __device__ inline void saveData(T * outReal, T * outImag,
//         const T resultReal, const T resultImag, const int numBasis,
//         const size_t m, const size_t n, const size_t p, const size_t q,
//         const int quad, const size_t blockPos, const T multiplier,
//         T sSumsReal[], T sSumsImag[]) {
//   if(quad == 1) {
//     outReal[m*numBasis + n] = resultReal * multiplier;
//     outImag[m*numBasis + n] = resultImag * multiplier;
// // TODO
// //   } else if(quad == 4) {
// //     __syncthreads();
// //     if(p < 2) {
// //       sSumsReal[blockPos] += sSumsReal[blockPos + 2];
// //       sSumsImag[blockPos] += sSumsImag[blockPos + 2];
// //     }
// //     __syncthreads();
// //     if(p == 0) {
// //       sSumsReal[blockPos] += sSumsReal[blockPos + 1];
// //       sSumsImag[blockPos] += sSumsImag[blockPos + 1];
// //     }
// //     __syncthreads();
// //     if(q < 2) {
// //       sSumsReal[blockPos] += sSumsReal[blockPos + (2*blockDim.x)];
// //       sSumsImag[blockPos] += sSumsImag[blockPos + (2*blockDim.x)];
// //     }
// //     __syncthreads();
// //     if(q == 0) {
// //       sSumsReal[blockPos] += sSumsReal[blockPos + blockDim.x];
// //       sSumsImag[blockPos] += sSumsImag[blockPos + blockDim.x];
// //       if(p == 0) {
// //         outReal[m*numBasis + n] = sSumsReal[blockPos] * multiplier;
// //         outImag[m*numBasis + n] = sSumsImag[blockPos] * multiplier;
// //       }
// //     }
// //   } else if(quad == 16) {
// //     __syncthreads();
// //     if(p < 8) {
// //       sSumsReal[blockPos] += sSumsReal[blockPos + 8];
// //       sSumsImag[blockPos] += sSumsImag[blockPos + 8];
// //     }
// //     __syncthreads();
// //     if(p < 4) {
// //       sSumsReal[blockPos] += sSumsReal[blockPos + 4];
// //       sSumsImag[blockPos] += sSumsImag[blockPos + 4];
// //     }
// //     __syncthreads();
// //     if(p < 2) {
// //       sSumsReal[blockPos] += sSumsReal[blockPos + 2];
// //       sSumsImag[blockPos] += sSumsImag[blockPos + 2];
// //     }
// //     __syncthreads();
// //     if(p == 0) {
// //       sSumsReal[blockPos] += sSumsReal[blockPos + 1];
// //       sSumsImag[blockPos] += sSumsImag[blockPos + 1];
// //     }
// //     __syncthreads();
// //     if(q < 8) {
// //       sSumsReal[blockPos] += sSumsReal[blockPos + (8*blockDim.x)];
// //       sSumsImag[blockPos] += sSumsImag[blockPos + (8*blockDim.x)];
// //     }
// //     __syncthreads();
// //     if(q < 4) {
// //       sSumsReal[blockPos] += sSumsReal[blockPos + (4*blockDim.x)];
// //       sSumsImag[blockPos] += sSumsImag[blockPos + (4*blockDim.x)];
// //     }
// //     __syncthreads();
// //     if(q < 2) {
// //       sSumsReal[blockPos] += sSumsReal[blockPos + (2*blockDim.x)];
// //       sSumsImag[blockPos] += sSumsImag[blockPos + (2*blockDim.x)];
// //     }
// //     __syncthreads();
// //     if(q == 0) {
// //       sSumsReal[blockPos] += sSumsReal[blockPos + blockDim.x];
// //       sSumsImag[blockPos] += sSumsImag[blockPos + blockDim.x];
// //       if(p == 0) {
// //         outReal[m*numBasis + n] = sSumsReal[blockPos] * multiplier;
// //         outImag[m*numBasis + n] = sSumsImag[blockPos] * multiplier;
// //       }
// //     }
//   } else {
//     CudaAtomicAdd<T> caa;    
//     caa.AtomicAdd(&outReal[m*numBasis + n], resultReal * multiplier);
//     caa.AtomicAdd(&outImag[m*numBasis + n], resultImag * multiplier);
//   }
// }

template<typename T>
__device__ inline void saveData(T * outReal, T * outImag,
        const T resultReal, const T resultImag, const int numBasis,
        const size_t m, const size_t n, const int quad, const T multiplier) {
  if(quad == 1) {
    outReal[m*numBasis + n] = resultReal * multiplier;
    outImag[m*numBasis + n] = resultImag * multiplier;
  } else {
    CudaAtomicAdd<T> caa;    
    caa.AtomicAdd(&outReal[m*numBasis + n], resultReal * multiplier);
    caa.AtomicAdd(&outImag[m*numBasis + n], resultImag * multiplier);
  }
}
#endif