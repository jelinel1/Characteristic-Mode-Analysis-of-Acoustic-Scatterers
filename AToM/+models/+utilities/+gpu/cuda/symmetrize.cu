
#include <hip/hip_runtime.h>
template <typename T>
__device__ void symmetrize(T * outMat, const int numBasisFcns) {
  size_t const x = blockDim.x * blockIdx.x + threadIdx.x;
  size_t const y = blockDim.y * blockIdx.y + threadIdx.y;

  if(x >= numBasisFcns || y >= numBasisFcns) {
    return;
  }
   
  if(x > y) {
    T tmp = (outMat[x*numBasisFcns + y] + outMat[y*numBasisFcns + x])/2;
    outMat[x*numBasisFcns + y] = tmp;
    outMat[y*numBasisFcns + x] = tmp;
  }
}

__global__ void symmetrizef(float * outMat, const int numBasisFcns) {
   symmetrize(outMat, numBasisFcns);
}

__global__ void symmetrized(double * outMat, const int numBasisFcns) {
   symmetrize(outMat, numBasisFcns);
}