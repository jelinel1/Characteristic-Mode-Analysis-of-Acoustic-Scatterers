#include "hip/hip_runtime.h"
#pragma once
#ifndef CUDA_MATHEMATICAL_FUNCTIONS_H
#define CUDA_MATHEMATICAL_FUNCTIONS_H

struct MathematicalFunc {
   template <typename T>
   static T cudaCos(T);
   
   template <typename T>
   static T cudaSin(T);

   template <typename T>
   static T cudaSqrt(T);

   template <typename T>
   static T cudaLog(T); // natural logarithm
           
   template <typename T>
   static T cudaAbs(T);

   template <typename T>
   static T cudaAtan(T);
};

template <typename T>
T MathematicalFunc::cudaCos(T) {
   extern __device__ void error(void);
   error(); // Ensure that we won't compile any un-specialized types
   return NULL;
}

template <typename T>
T MathematicalFunc::cudaSin(T) {
   extern __device__ void error(void);
   error(); // Ensure that we won't compile any un-specialized types
   return NULL;
}

template <typename T>
T MathematicalFunc::cudaSqrt(T) {
   extern __device__ void error(void);
   error(); // Ensure that we won't compile any un-specialized types
   return NULL;
}

template <typename T>
T MathematicalFunc::cudaLog(T) {
   extern __device__ void error(void);
   error(); // Ensure that we won't compile any un-specialized types
   return NULL;
}

template <typename T>
T MathematicalFunc::cudaAbs(T) {
   extern __device__ void error(void);
   error(); // Ensure that we won't compile any un-specialized types
   return NULL;
}

template <typename T>
T MathematicalFunc::cudaAtan(T) {
   extern __device__ void error(void);
   error(); // Ensure that we won't compile any un-specialized types
   return NULL;
}

template <>
__device__ inline float MathematicalFunc::cudaCos(float value) {
  return cosf(value);
}

template <>
__device__ inline double MathematicalFunc::cudaCos(double value) {
  return cos(value);
}

template <>
__device__ inline float MathematicalFunc::cudaSin(float value) {
  return sinf(value);
}

template <>
__device__ inline double MathematicalFunc::cudaSin(double value) {
  return sin(value);
}

template <>
__device__ inline float MathematicalFunc::cudaSqrt(float value) {
  return sqrtf(value);
}

template <>
__device__ inline double MathematicalFunc::cudaSqrt(double value) {
  return sqrt(value);
}

template <>
__device__ inline float MathematicalFunc::cudaLog(float value) {
  return logf(value);
}

template <>
__device__ inline double MathematicalFunc::cudaLog(double value) {
  return log(value);
}

template <>
__device__ inline float MathematicalFunc::cudaAbs(float value) {
  return fabsf(value);
}

template <>
__device__ inline double MathematicalFunc::cudaAbs(double value) {
  return fabs(value);
}

template <>
__device__ inline float MathematicalFunc::cudaAtan(float value) {
  return atanf(value);
}

template <>
__device__ inline double MathematicalFunc::cudaAtan(double value) {
  return atan(value);
}
#endif // !CUDA_MATHEMATICAL_FUNCTIONS_H