#pragma once
#include <hip/hip_runtime.h>

#ifndef CUDA_ATOMIC_ADD_TEMPLATE_H
#define CUDA_ATOMIC_ADD_TEMPLATE_H

template <typename T>
struct CudaAtomicAdd
{
	__device__ T AtomicAdd(T* ref, T value)
	{
		extern __device__ void error(void);
		error(); // Ensure that we won't compile any un-specialized types
		return NULL;
	}
};
template <>
struct CudaAtomicAdd <int>
{
	__device__ unsigned int AtomicAdd(int* ref, int value)
	{
		return atomicAdd(ref, value);
	}
};
template <>
struct CudaAtomicAdd <unsigned int>
{
	__device__ unsigned int AtomicAdd(unsigned int* ref, unsigned int value)
	{
		return atomicAdd(ref, value);
	}
};
template <>
struct CudaAtomicAdd <unsigned long long int>
{
	__device__ unsigned int AtomicAdd(unsigned long long int* ref, unsigned long long int value)
	{
		return atomicAdd(ref, value);
	}
};
template <>
struct CudaAtomicAdd <float>
{
	__device__ float AtomicAdd(float* ref, float value)
	{
		return atomicAdd(ref, value);
	}
};
template <>
struct CudaAtomicAdd <double>
{
	__device__ double AtomicAdd(double* ref, double value)
	{
    	// double is different becase it is only supported in later architectures
	#if __CUDA_ARCH__ < 600
		unsigned long long int* address_as_ull = (unsigned long long int*)ref;
		unsigned long long int old = *address_as_ull, assumed;
		do {
			assumed = old;
			old = atomicCAS(address_as_ull, 
    			    assumed,
    			    __double_as_longlong(
    			        value + __longlong_as_double(assumed)
    			    )
    			);
		} while (assumed != old);
		return __longlong_as_double(old);
	#else
		return atomicAdd(ref, value);
	#endif
	}
};
#endif // !CUDA_ATOMIC_ADD_TEMPLATE_H