
#include <hip/hip_runtime.h>
template <typename T>
__device__ void transpose(T * outMat, const int numBasisFcns) {
  size_t const x = blockDim.x * blockIdx.x + threadIdx.x;
  size_t const y = blockDim.y * blockIdx.y + threadIdx.y;

  if(x >= numBasisFcns || y >= numBasisFcns) {
    return;
  }
   
  if(x > y) {
    outMat[x*numBasisFcns + y] = outMat[y*numBasisFcns + x];
  } 
}

__global__ void transposef(float * outMat, const int numBasisFcns) {
   transpose(outMat, numBasisFcns);
}

__global__ void transposed(double * outMat, const int numBasisFcns) {
   transpose(outMat, numBasisFcns);
}