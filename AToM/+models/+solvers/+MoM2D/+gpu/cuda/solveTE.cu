#include "hip/hip_runtime.h"
#include "../../../../+utilities/+gpu/cuda/helpers.cu"

template <typename T>
__device__ void solveTE(T * outReal, T * outImag, const T * weights,
        const T * radiusVectors, const T wavelength,
        const int quad, const T* edgeLengths, const T scale, const int xMin,
        const int xMax, const int yMin, const int yMax, const int outputSize) {
           
  size_t const x = blockDim.x * blockIdx.x + threadIdx.x + (xMin * quad);
  size_t const y = blockDim.y * blockIdx.y + threadIdx.y + (yMin * quad);

  if(x >= xMax * quad || y >= yMax * quad) {
     return;
  }

  size_t const m = floorf(x/quad);
  size_t const n = floorf(y/quad);

  // stop condition - compute only half of the symetric matrix
  if(m > n) {
     return;
  }

//   __shared__ T sSumsReal[1024];
//   __shared__ T sSumsImag[1024];
//   size_t const blockPos = threadIdx.y*blockDim.x + threadIdx.x;

  size_t const p = x % quad;
  size_t const q = y % quad;

  const T weight = weights[p*quad + q];

  const T* radius1 = &radiusVectors[m*quad*2*3]; // m-th radius function, each represented by 2*quad points, each point in 3D system
  const T* radius2 = &radiusVectors[n*quad*2*3];

  T resultReal = 0, resultImag = 0, dist;

  dist = computeDist(&radius1[q*3], &radius2[p*3]);
  positiveXY(-wavelength*dist, weight, resultReal, resultImag);

  dist = computeDist(&radius1[(q+quad)*3], &radius2[p*3]);
  negativeXY(-wavelength*dist, weight, resultReal, resultImag);

  dist = computeDist(&radius1[q*3], &radius2[(p+quad)*3]);
  negativeXY(-wavelength*dist, weight, resultReal, resultImag);

  dist = computeDist(&radius1[(q+quad)*3], &radius2[(p+quad)*3]);
  positiveXY(-wavelength*dist, weight, resultReal, resultImag);

//   sSumsReal[blockPos] = resultReal;
//   sSumsImag[blockPos] = resultImag;

  const T multiplier = (edgeLengths[m] * edgeLengths[n]) / scale;
//   saveData(outReal, outImag, resultReal, resultImag, outputSize, m, n, p, q,
//            quad, blockPos, multiplier, sSumsReal, sSumsImag);
  saveData(outReal, outImag, resultReal, resultImag, outputSize,
           m - xMin, n - yMin, quad, multiplier);
}

__global__ void solveTEf(float * outReal, float * outImag, const float * weights,
        const float * radiusVectors, const float wavelength, const int quad,
        const float* edgeLengths, const float scale, const int xMin,
        const int xMax, const int yMin, const int yMax, const int outputSize) {
  solveTE(outReal, outImag, weights, radiusVectors, wavelength,
          quad, edgeLengths, scale, xMin, xMax, yMin, yMax, outputSize);
}

__global__ void solveTEd(double * outReal, double * outImag, const double * weights,
        const double * radiusVectors, const double wavelength, const int quad,
        const double* edgeLengths, const double scale, const int xMin,
        const int xMax, const int yMin, const int yMax, const int outputSize) {
  solveTE(outReal, outImag, weights, radiusVectors, wavelength,
          quad, edgeLengths, scale, xMin, xMax, yMin, yMax, outputSize);
}
