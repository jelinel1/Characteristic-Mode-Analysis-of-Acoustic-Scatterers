#include "hip/hip_runtime.h"
#include "ITT.cu"

template <typename T>
__device__ void solveZE0(T* outReal,
        const T* edgeLengths, const int* triaPlus, const int* triaMinus, const T scale,
        const T* p1, const T* p2, const T* p3, const T* xVec, const T* yVec, const T* zVec,
        const T* w, const T* s1, const T* s2, const T* s3,
        const T* m1, const T* m2, const T* m3,
        const int quad, const T* weight, const T* triaArea, const int xMin,
        const int xMax, const int yMin, const int yMax, const int outputSize) {
           
  size_t const x = blockDim.x * blockIdx.x + threadIdx.x + xMin;
  size_t const y = blockDim.y * blockIdx.y + threadIdx.y + yMin;
  
  if(x >= xMax || y >= yMax) {
     return;
  }

  const int dim = 3;
  int m = triaPlus[x], n = triaPlus[y];
  T sum = 0, itt3;

  for(int i = 0; i < quad; i++) {
    itt3 = ITT3(&p1[m*dim], &p2[m*dim], &p3[m*dim], xVec[n*quad + i], yVec[n*quad + i], zVec[n*quad + i], &w[m*dim],
              &s1[m*dim], &s2[m*dim], &s3[m*dim], &m1[m*dim], &m2[m*dim], &m3[m*dim], weight[i]);
    sum += itt3 / triaArea[m];
  }
  
  m = triaPlus[x], n = triaMinus[y];
  for(int i = 0; i < quad; i++) {
    itt3 = ITT3(&p1[m*dim], &p2[m*dim], &p3[m*dim], xVec[n*quad + i], yVec[n*quad + i], zVec[n*quad + i], &w[m*dim],
             &s1[m*dim], &s2[m*dim], &s3[m*dim], &m1[m*dim], &m2[m*dim], &m3[m*dim], weight[i]);
    sum -= itt3 / triaArea[m];
  }

  m = triaMinus[x], n = triaPlus[y];
  for(int i = 0; i < quad; i++) {
    itt3 = ITT3(&p1[m*dim], &p2[m*dim], &p3[m*dim], xVec[n*quad + i], yVec[n*quad + i], zVec[n*quad + i], &w[m*dim],
             &s1[m*dim], &s2[m*dim], &s3[m*dim], &m1[m*dim], &m2[m*dim], &m3[m*dim], weight[i]);
    sum -= itt3 / triaArea[m];
  }

  m = triaMinus[x], n = triaMinus[y];
  for(int i = 0; i < quad; i++) {
    itt3 = ITT3(&p1[m*dim], &p2[m*dim], &p3[m*dim], xVec[n*quad + i], yVec[n*quad + i], zVec[n*quad + i], &w[m*dim],
             &s1[m*dim], &s2[m*dim], &s3[m*dim], &m1[m*dim], &m2[m*dim], &m3[m*dim], weight[i]);
    sum += itt3 / triaArea[m];
  }

  outReal[(x - xMin) + (y - yMin)*outputSize] = (edgeLengths[x] * edgeLengths[y] * sum) / scale;
}

__global__ void solveZE0f(float* outReal,
        const float* edgeLengths, const int* triaPlus, const int* triaMinus, const float scale,
        const float* p1, const float* p2, const float* p3, const float* x, const float* y, const float* z,
        const float* w, const float* s1, const float* s2, const float* s3,
        const float* m1, const float* m2, const float* m3,
        const int quad, const float* weight, const float* triaArea, const int xMin,
        const int xMax, const int yMin, const int yMax, const int outputSize) {
  solveZE0(outReal, edgeLengths, triaPlus, triaMinus, scale,
        p1, p2, p3, x, y, z, w, s1, s2, s3, m1, m2, m3, quad,
        weight, triaArea, xMin, xMax, yMin, yMax, outputSize);
}

__global__ void solveZE0d(double* outReal,
        const double* edgeLengths, const int* triaPlus, const int* triaMinus, const double scale,
        const double* p1, const double* p2, const double* p3, const double* x, const double* y, const double* z,
        const double* w, const double* s1, const double* s2, const double* s3,
        const double* m1, const double* m2, const double* m3,
        const int quad, const double* weight, const double* triaArea, const int xMin,
        const int xMax, const int yMin, const int yMax, const int outputSize) {
  solveZE0(outReal, edgeLengths, triaPlus, triaMinus, scale,
        p1, p2, p3, x, y, z, w, s1, s2, s3, m1, m2, m3, quad,
        weight, triaArea, xMin, xMax, yMin, yMax, outputSize);
}