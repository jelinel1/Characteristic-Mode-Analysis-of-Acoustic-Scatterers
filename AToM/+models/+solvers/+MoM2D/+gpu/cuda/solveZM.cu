#include "hip/hip_runtime.h"
#include "../../../../+utilities/+gpu/cuda/helpers.cu"

template <typename T>
__device__ void solveZM(T * outReal, T * outImag, const T * weights,
        const T * radiusVectors, const T * basisVectors, const T wavelength,
        const int quad, const T* edgeLengths, const T scale, const int xMin,
        const int xMax, const int yMin, const int yMax, const int outputSize) {
           
  size_t const x = blockDim.x * blockIdx.x + threadIdx.x + (xMin * quad);
  size_t const y = blockDim.y * blockIdx.y + threadIdx.y + (yMin * quad);
  
  if(x >= xMax * quad || y >= yMax * quad) {
     return;
  }

  size_t const m = floorf(x/quad);
  size_t const n = floorf(y/quad);

  // stop condition - compute only half of the symetric matrix 
  if(m > n) {
     return;
  }

//   __shared__ T sSumsReal[1024];
//   __shared__ T sSumsImag[1024];
//   size_t const blockPos = threadIdx.y*blockDim.x + threadIdx.x;

  size_t const p = x % quad;
  size_t const q = y % quad;

  T weight = weights[p*quad + q];

  const T* radius1 = &radiusVectors[m*quad*2*3]; // m-th radius function, each represented by 2*quad points, each point in 3D system
  const T* radius2 = &radiusVectors[n*quad*2*3];

  const T* basis1 = &basisVectors[m*quad*2*3];
  const T* basis2 = &basisVectors[n*quad*2*3];

  T resultReal = 0, resultImag = 0, dist, basisDotProduct;

  dist = computeDist(&radius1[q*3], &radius2[p*3]);
  basisDotProduct = dotProduct3(&basis1[q*3], &basis2[p*3]);
  positiveXYDivided(-wavelength*dist, weight*basisDotProduct, resultReal,
                     resultImag, dist, wavelength);

  dist = computeDist(&radius1[(q+quad)*3], &radius2[p*3]);
  basisDotProduct = dotProduct3(&basis1[(q+quad)*3], &basis2[p*3]);
  negativeXYDivided(-wavelength*dist, weight*basisDotProduct, resultReal,
                     resultImag, dist, wavelength);

  dist = computeDist(&radius1[q*3], &radius2[(p+quad)*3]);
  basisDotProduct = dotProduct3(&basis1[q*3], &basis2[(p+quad)*3]);
  negativeXYDivided(-wavelength*dist, weight*basisDotProduct, resultReal,
                     resultImag, dist, wavelength);

  dist = computeDist(&radius1[(q+quad)*3], &radius2[(p+quad)*3]);
  basisDotProduct = dotProduct3(&basis1[(q+quad)*3], &basis2[(p+quad)*3]);
  positiveXYDivided(-wavelength*dist, weight*basisDotProduct, resultReal,
                     resultImag, dist, wavelength);

//   sSumsReal[blockPos] = resultReal;
//   sSumsImag[blockPos] = resultImag;

  const T multiplier = (edgeLengths[m] * edgeLengths[n]) / scale;
//   saveData(outReal, outImag, resultReal, resultImag, numBasisFcns, m, n, p, q,
//            quad, blockPos, multiplier, sSumsReal, sSumsImag);
  saveData(outReal, outImag, resultReal, resultImag, outputSize,
           m - xMin, n - yMin, quad, multiplier);
}

__global__ void solveZMf(float * outReal, float * outImag, const float * weights,
        const float * radiusVectors, const float * basisVectors, const float wavelength,
        const int quad, const float* edgeLengths, const float scale, const int xMin,
        const int xMax, const int yMin, const int yMax, const int outputSize) {
  solveZM(outReal, outImag, weights, radiusVectors, basisVectors, wavelength,
          quad, edgeLengths, scale, xMin, xMax, yMin, yMax, outputSize);
}

__global__ void solveZMd(double * outReal, double * outImag, const double * weights,
        const double * radiusVectors, const double * basisVectors, const double wavelength,
        const int quad, const double* edgeLengths, const double scale, const int xMin,
        const int xMax, const int yMin, const int yMax, const int outputSize) {
  solveZM(outReal, outImag, weights, radiusVectors, basisVectors, wavelength,
          quad, edgeLengths, scale, xMin, xMax, yMin, yMax, outputSize);
}