#include "hip/hip_runtime.h"
#include "ITT.cu"

template <typename T>
__device__ void solveZM0(T* outReal,
        const T* edgeLengths, const int* triaPlus, const int* triaMinus, const T scale,
        const T* p1, const T* p2, const T* p3, const T* xVec, const T* yVec, const T* zVec,
        const T* u, const T* v, const T* w, const T* s1, const T* s2, const T * s3,
        const T* m1, const T* m2, const T* m3, const int quad,
        const T* weight, const T* triaArea, const T* rPlus, const T* rMinus, const int xMin,
        const int xMax, const int yMin, const int yMax, const int outputSize) {
  
  size_t const x = blockDim.x * blockIdx.x + threadIdx.x + xMin;
  size_t const y = blockDim.y * blockIdx.y + threadIdx.y + yMin;
  
  if(x >= xMax || y >= yMax) {
     return;
  }

  const int dim = 3;
  T tmp_pp[dim] = { 0 }, tmp_mp[dim] = { 0 }, tmp_pm[dim] = { 0 }, tmp_mm[dim] = { 0 };
  int m = triaPlus[y], n = triaPlus[x];
  T ITT1 = 0, ITT3 = 0;
  T sum = 0;

  for(int i = 0; i < quad; i++) {
    ITT(ITT1, tmp_pp[0], tmp_pp[1], tmp_pp[2], ITT3, &p1[n*dim], &p2[n*dim], &p3[n*dim], xVec[m*quad + i],
        yVec[m*quad + i], zVec[m*quad + i], &u[n*dim], &v[n*dim], &w[n*dim],
        &s1[n*dim], &s2[n*dim], &s3[n*dim], &m1[n*dim], &m2[n*dim], &m3[n*dim], weight[i], triaArea[n]);
  }
  sum += ITT1;
  sum += dotProduct3(&rPlus[y*dim], &rPlus[x*dim]) * ITT3;
  sum -= dotProduct3Static(tmp_pp, &rPlus[x*dim]);
  
  m = triaMinus[y], n = triaPlus[x];
  ITT1 = 0, ITT3 = 0;
  for(int i = 0; i < quad; i++) {
    ITT(ITT1, tmp_mp[0], tmp_mp[1], tmp_mp[2], ITT3, &p1[n*dim], &p2[n*dim], &p3[n*dim], xVec[m*quad + i],
        yVec[m*quad + i], zVec[m*quad + i], &u[n*dim], &v[n*dim], &w[n*dim],
        &s1[n*dim], &s2[n*dim], &s3[n*dim], &m1[n*dim], &m2[n*dim], &m3[n*dim], weight[i], triaArea[n]);
  }
  sum -= ITT1;
  sum -= dotProduct3(&rMinus[y*dim], &rPlus[x*dim]) * ITT3;
  sum += dotProduct3Static(tmp_mp, &rPlus[x*dim]);
  
  m = triaPlus[y], n = triaMinus[x];
  ITT1 = 0, ITT3 = 0;
  for(int i = 0; i < quad; i++) {
    ITT(ITT1, tmp_pm[0], tmp_pm[1], tmp_pm[2], ITT3, &p1[n*dim], &p2[n*dim], &p3[n*dim], xVec[m*quad + i],
        yVec[m*quad + i], zVec[m*quad + i], &u[n*dim], &v[n*dim], &w[n*dim],
        &s1[n*dim], &s2[n*dim], &s3[n*dim], &m1[n*dim], &m2[n*dim], &m3[n*dim], weight[i], triaArea[n]);
  }
  sum -= ITT1;
  sum -= dotProduct3(&rMinus[x*dim], &rPlus[y*dim]) * ITT3;
  sum += dotProduct3Static(tmp_pm, &rMinus[x*dim]);

  m = triaMinus[y], n = triaMinus[x];
  ITT1 = 0, ITT3 = 0;
  for(int i = 0; i < quad; i++) {
    ITT(ITT1, tmp_mm[0], tmp_mm[1], tmp_mm[2], ITT3, &p1[n*dim], &p2[n*dim], &p3[n*dim], xVec[m*quad + i],
        yVec[m*quad + i], zVec[m*quad + i], &u[n*dim], &v[n*dim], &w[n*dim],
        &s1[n*dim], &s2[n*dim], &s3[n*dim], &m1[n*dim], &m2[n*dim], &m3[n*dim], weight[i], triaArea[n]);
  }
  sum += ITT1;
  sum += dotProduct3(&rMinus[y*dim], &rMinus[x*dim]) * ITT3;
  sum -= dotProduct3Static(tmp_mm, &rMinus[x*dim]);

  /*---------------------------------*/
      
  tmp_pp[0] = 0;
  tmp_pp[1] = 0;
  tmp_pp[2] = 0;
  m = triaPlus[x], n = triaPlus[y];
  for(int i = 0; i < quad; i++) {
    ITT(ITT1, tmp_pp[0], tmp_pp[1], tmp_pp[2], ITT3, &p1[n*dim], &p2[n*dim], &p3[n*dim], xVec[m*quad + i],
        yVec[m*quad + i], zVec[m*quad + i], &u[n*dim], &v[n*dim], &w[n*dim],
        &s1[n*dim], &s2[n*dim], &s3[n*dim], &m1[n*dim], &m2[n*dim], &m3[n*dim], weight[i], triaArea[n]);
  }
  sum -= dotProduct3Static(tmp_pp, &rPlus[y*dim]);

  tmp_mp[0] = 0;
  tmp_mp[1] = 0;
  tmp_mp[2] = 0;
  m = triaMinus[x], n = triaPlus[y];
  for(int i = 0; i < quad; i++) {
    ITT(ITT1, tmp_mp[0], tmp_mp[1], tmp_mp[2], ITT3, &p1[n*dim], &p2[n*dim], &p3[n*dim], xVec[m*quad + i],
        yVec[m*quad + i], zVec[m*quad + i], &u[n*dim], &v[n*dim], &w[n*dim],
        &s1[n*dim], &s2[n*dim], &s3[n*dim], &m1[n*dim], &m2[n*dim], &m3[n*dim], weight[i], triaArea[n]);
  }
  sum += dotProduct3Static(tmp_mp, &rPlus[y*dim]);

  tmp_pm[0] = 0;
  tmp_pm[1] = 0;
  tmp_pm[2] = 0;
  m = triaPlus[x], n = triaMinus[y];
  for(int i = 0; i < quad; i++) {
    ITT(ITT1, tmp_pm[0], tmp_pm[1], tmp_pm[2], ITT3, &p1[n*dim], &p2[n*dim], &p3[n*dim], xVec[m*quad + i],
        yVec[m*quad + i], zVec[m*quad + i], &u[n*dim], &v[n*dim], &w[n*dim],
        &s1[n*dim], &s2[n*dim], &s3[n*dim], &m1[n*dim], &m2[n*dim], &m3[n*dim], weight[i], triaArea[n]);
  }
  sum += dotProduct3Static(tmp_pm, &rMinus[y*dim]);

  tmp_mm[0] = 0;
  tmp_mm[1] = 0;
  tmp_mm[2] = 0;
  m = triaMinus[x], n = triaMinus[y];
  for(int i = 0; i < quad; i++) {
    ITT(ITT1, tmp_mm[0], tmp_mm[1], tmp_mm[2], ITT3, &p1[n*dim], &p2[n*dim], &p3[n*dim], xVec[m*quad + i],
        yVec[m*quad + i], zVec[m*quad + i], &u[n*dim], &v[n*dim], &w[n*dim],
        &s1[n*dim], &s2[n*dim], &s3[n*dim], &m1[n*dim], &m2[n*dim], &m3[n*dim], weight[i], triaArea[n]);
  }
  sum -= dotProduct3Static(tmp_mm, &rMinus[y*dim]);

  outReal[(x - xMin) + (y - yMin)*outputSize] = (edgeLengths[x] * edgeLengths[y] * sum) / scale;
}

__global__ void solveZM0f(float* outReal,
        const float* edgeLengths, const int* triaPlus, const int* triaMinus, const float scale,
        const float* p1, const float* p2, const float* p3, const float* x, const float* y, const float* z,
        const float* u, const float* v, const float* w, const float* s1, const float* s2, const float* s3,
        const float* m1, const float* m2, const float* m3, const int quad,
        const float* weight, const float* triaArea, const float* rPlus, const float* rMinus, const int xMin,
        const int xMax, const int yMin, const int yMax, const int outputSize) {
  solveZM0(outReal, edgeLengths, triaPlus, triaMinus, scale,
        p1, p2, p3, x, y, z, u, v, w, s1, s2, s3, m1, m2, m3,
        quad, weight, triaArea, rPlus, rMinus, xMin, xMax, yMin, yMax, outputSize);
}

__global__ void solveZM0d(double* outReal,
        const double* edgeLengths, const int* triaPlus, const int* triaMinus, const double scale,
        const double* p1, const double* p2, const double* p3, const double* x, const double* y, const double* z,
        const double* u, const double* v, const double* w, const double* s1, const double* s2, const double* s3,
        const double* m1, const double* m2, const double* m3, const int quad,
        const double* weight, const double* triaArea, const double* rPlus, const double* rMinus, const int xMin,
        const int xMax, const int yMin, const int yMax, const int outputSize) {
  solveZM0(outReal, edgeLengths, triaPlus, triaMinus, scale,
        p1, p2, p3, x, y, z, u, v, w, s1, s2, s3, m1, m2, m3,
        quad, weight, triaArea, rPlus, rMinus, xMin, xMax, yMin, yMax, outputSize);
}