#include "hip/hip_runtime.h"
#include "../../../../+utilities/+gpu/cuda/helpers.cu"

template <typename T>
__device__ void ITT(T &ITT1, T &ITT2x, T &ITT2y, T &ITT2z, T &ITT3,
        const T * p1, const T * p2, const T * p3, const T x, const T y, const T z,
        const T * u, const T * v, const T * w,  const T * s1, const T * s2, const T * s3,
        const T * m1, const T * m2, const T * m3, const T weight, const T triaArea) {
  const int dim = 3;
  T R1[dim], R2[dim], R3[dim], quadCoord[dim], tmp[dim];

  quadCoord[0] = x;
  quadCoord[1] = y;
  quadCoord[2] = z;
  vectorMinus3(R1, p1, quadCoord);
  vectorMinus3(R2, p2, quadCoord);
  vectorMinus3(R3, p3, quadCoord);

  T t10 = dotProduct3Static(R2, m1);
  T t20 = dotProduct3Static(R3, m2);
  T t30 = dotProduct3Static(R1, m3);

  T ww = dotProduct3Static(R1, w);

  T R10 = hypot(t10, ww);
  T R20 = hypot(t20, ww);
  T R30 = hypot(t30, ww);

  T R1n = norm2(R2);
  T R1p = norm2(R3);
  T R2p = norm2(R1);

  T s1n = dotProduct3Static(R2, s1);
  T s1p = dotProduct3Static(R3, s1);
  T s2n = dotProduct3Static(R3, s2);
  T s2p = dotProduct3Static(R1, s2);
  T s3n = dotProduct3Static(R1, s3);
  T s3p = dotProduct3Static(R2, s3);
  T I11 = RWGl10(s1n, s1p, R1n, R1p);
  T I21 = RWGl10(s2n, s2p, R1p, R2p);
  T I31 = RWGl10(s3n, s3p, R2p, R1n);

  vectorMinus3(tmp, quadCoord, p1);
  T w0 = dotProduct3(tmp, w);

  T beta = 0;
  beta += RWGs00(s1n, s1p, t10, w0, R10, R1n, R1p);
  beta += RWGs00(s2n, s2p, t20, w0, R20, R1p, R2p);
  beta += RWGs00(s3n, s3p, t30, w0, R30, R2p, R1n);

  T S = t10*I11 + t20*I21 + t30*I31 - MathematicalFunc::cudaAbs(w0)*beta;

  T I12 = RWG111(s1n, s1p, R10, R1n, R1p, I11);
  T I22 = RWG111(s2n, s2p, R20, R1p, R2p, I21);
  T I32 = RWG111(s3n, s3p, R30, R2p, R1n, I31);

  T Su = dotProduct3(u, m1)*I12 + dotProduct3(u, m2)*I22 + dotProduct3(u, m3)*I32 + dotProduct3(tmp, u)*S;
  T Sv = dotProduct3(v, m1)*I12 + dotProduct3(v, m2)*I22 + dotProduct3(v, m3)*I32 + dotProduct3(tmp, v)*S;

  T ITT1tmp = quadCoord[0] * (Su*u[0] + Sv*v[0] + S*p1[0])
            + quadCoord[1] * (Su*u[1] + Sv*v[1] + S*p1[1])
            + quadCoord[2] * (Su*u[2] + Sv*v[2] + S*p1[2]);

  ITT1 += (2 * ITT1tmp * weight) / triaArea;
  ITT2x += (2 * S * weight * quadCoord[0]) / triaArea;
  ITT2y += (2 * S * weight * quadCoord[1]) / triaArea;
  ITT2z += (2 * S * weight * quadCoord[2]) / triaArea;
  ITT3 += (2 * S * weight) / triaArea;
}

template <typename T>
__device__ T ITT3(const T * p1, const T * p2, const T * p3, const T x, const T y, const T z,
        const T * w,  const T * s1, const T * s2, const T * s3,
        const T * m1, const T * m2, const T * m3, const T weight) {
  const int dim = 3;
  T R1[dim], R2[dim], R3[dim], quadCoord[dim], tmp[dim];

  quadCoord[0] = x;
  quadCoord[1] = y;
  quadCoord[2] = z;
  vectorMinus3(R1, p1, quadCoord);
  vectorMinus3(R2, p2, quadCoord);
  vectorMinus3(R3, p3, quadCoord);

  T t10 = dotProduct3Static(R2, m1);
  T t20 = dotProduct3Static(R3, m2);
  T t30 = dotProduct3Static(R1, m3);

  T ww = dotProduct3Static(R1, w);

  T R10 = hypot(t10, ww);
  T R20 = hypot(t20, ww);
  T R30 = hypot(t30, ww);

  T R1n = norm2(R2);
  T R1p = norm2(R3);
  T R2p = norm2(R1);

  T s1n = dotProduct3Static(R2, s1);
  T s1p = dotProduct3Static(R3, s1);
  T s2n = dotProduct3Static(R3, s2);
  T s2p = dotProduct3Static(R1, s2);
  T s3n = dotProduct3Static(R1, s3);
  T s3p = dotProduct3Static(R2, s3);
  T I11 = RWGl10(s1n, s1p, R1n, R1p);
  T I21 = RWGl10(s2n, s2p, R1p, R2p);
  T I31 = RWGl10(s3n, s3p, R2p, R1n);

  vectorMinus3(tmp, quadCoord, p1);
  T w0 = dotProduct3(tmp, w);

  T beta = 0;
  beta += RWGs00(s1n, s1p, t10, w0, R10, R1n, R1p);
  beta += RWGs00(s2n, s2p, t20, w0, R20, R1p, R2p);
  beta += RWGs00(s3n, s3p, t30, w0, R30, R2p, R1n);

  T S = t10*I11 + t20*I21 + t30*I31 - MathematicalFunc::cudaAbs(w0)*beta;
  return 2 * S * weight;
}