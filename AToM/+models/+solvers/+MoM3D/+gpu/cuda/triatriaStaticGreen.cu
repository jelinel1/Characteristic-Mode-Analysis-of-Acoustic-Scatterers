#include "hip/hip_runtime.h"
#include "../../../../+utilities/+gpu/cuda/helpers.cu"
#include "../../../+MoM2D/+gpu/cuda/ITT.cu"

template <typename T>
__device__ void triatriaStaticGreen(T* outReal, const T* p1,
        const T* p2, const T* p3, const T* x, const T* y, const T* z,
        const T* u, const T* v, const T* w, const T* s1, const T* s2, const T* s3,
        const T* m1, const T* m2, const T* m3, const T* weights, 
        const T* triArea, const int quad, const int xMin,
        const int xMax, const int yMin, const int yMax, const int outputDim) {
           
  size_t const m = blockDim.x * blockIdx.x + threadIdx.x + (xMin * quad);
  size_t const n = blockDim.y * blockIdx.y + threadIdx.y + yMin;

  if(m >= xMax * quad || n >= yMax) {
     return;
  }

  size_t const mTria = n;
  size_t const nTria = floorf(m / quad);
  size_t const nQuad = m % quad;

  const int dim = 3;
  T result = ITT3(&p1[mTria*dim], &p2[mTria*dim], &p3[mTria*dim],
          x[nTria*quad + nQuad], y[nTria*quad + nQuad], z[nTria*quad + nQuad],
          &w[mTria*dim], &s1[mTria*dim], &s2[mTria*dim], &s3[mTria*dim],
          &m1[mTria*dim], &m2[mTria*dim], &m3[mTria*dim], weights[nQuad]);

  CudaAtomicAdd<T> caa;    
  caa.AtomicAdd(&outReal[(mTria - yMin)*outputDim + (nTria - xMin)],
          result * triArea[nTria]);
}

__global__ void triatriaStaticGreenf(float* outReal, const float* p1,
        const float* p2, const float* p3, const float* x, const float* y, const float* z,
        const float* u, const float* v, const float* w, const float* s1, const float* s2, const float* s3,
        const float* m1, const float* m2, const float* m3, const float* weights,
        const float* triArea, const int quad,
        const int xMin, const int xMax, const int yMin, const int yMax, const int outputDim) {
  triatriaStaticGreen(outReal, p1, p2, p3, x, y, z, u, v, w, s1, s2, s3,
        m1, m2, m3, weights, triArea, quad, xMin, xMax, yMin, yMax, outputDim);
}

__global__ void triatriaStaticGreend(double* outReal, const double* p1,
        const double* p2, const double* p3, const double* x, const double* y, const double* z,
        const double* u, const double* v, const double* w, const double* s1, const double* s2, const double* s3,
        const double* m1, const double* m2, const double* m3, const double* weights,
        const double* triArea, const int quad,
        const int xMin, const int xMax, const int yMin, const int yMax, const int outputDim) {
  triatriaStaticGreen(outReal, p1, p2, p3, x, y, z, u, v, w, s1, s2, s3,
        m1, m2, m3, weights, triArea, quad, xMin, xMax, yMin, yMax, outputDim);
}
