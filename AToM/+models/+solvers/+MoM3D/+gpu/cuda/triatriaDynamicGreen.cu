#include "hip/hip_runtime.h"
#include "../../../../+utilities/+gpu/cuda/helpers.cu"

const double PI = 3.141592653589793238463;

template <typename T>
__device__ void triatriaDynamicGreen(T* outReal, T* outImag,
        const T* x, const T* y, const T* z, const T k,
        const T* weights, const T* triArea, const int quad,
        const int xMin, const int xMax,
        const int yMin, const int yMax, const int outputSize) {
  size_t const m = blockDim.x * blockIdx.x + threadIdx.x + (xMin * quad);
  size_t const n = blockDim.y * blockIdx.y + threadIdx.y + (yMin * quad);

  if(m >= xMax * quad || n >= yMax * quad) {
     return;
  }

  size_t const mTria = floorf(n / quad);
  size_t const nTria = floorf(m / quad);

  // stop condition - compute only half of the symetric matrix 
  if(mTria > nTria) {
     return;
  }

  size_t const mQuad = n % quad;
  size_t const nQuad = m % quad;

  const int dim = 3;
  T v1[dim], v2[dim];

  v1[0] = x[mTria*quad + mQuad];
  v1[1] = y[mTria*quad + mQuad];
  v1[2] = z[mTria*quad + mQuad];

  v2[0] = x[nTria*quad + nQuad];
  v2[1] = y[nTria*quad + nQuad];
  v2[2] = z[nTria*quad + nQuad];

  T kR = computeDist(v1, v2) * k;

  T resultReal, resultImag;
  if(kR < 1e-5) {
     resultReal = -kR / 2;
     resultImag = -1 + kR*kR / 6;
  } else {
     resultReal = (MathematicalFunc::cudaCos(kR) - 1) / kR;
     resultImag = - MathematicalFunc::cudaSin(kR) / kR;
  }

  T pi = PI;
  saveData(outReal, outImag, resultReal, resultImag,
           outputSize, mTria - yMin, nTria - xMin, quad,
           k * weights[mQuad] * weights[nQuad]
             * ((triArea[mTria] * triArea[nTria]) / pi));
}

__global__ void triatriaDynamicGreenf(float* outReal, float* outImag,
        const float* x, const float* y, const float* z, const float k,
        const float* weights, const float* triArea, const int quad,
        const int xMin, const int xMax, const int yMin, const int yMax, const int outputSize) {
  triatriaDynamicGreen(outReal, outImag, x, y, z, k, weights, triArea,
          quad, xMin, xMax, yMin, yMax, outputSize);
}

__global__ void triatriaDynamicGreend(double* outReal, double* outImag,
        const double* x, const double* y, const double* z, const double k,
        const double* weights, const double* triArea, const int quad,
        const int xMin, const int xMax, const int yMin, const int yMax, const int outputSize) {
  triatriaDynamicGreen(outReal, outImag, x, y, z, k, weights, triArea,
          quad, xMin, xMax, yMin, yMax, outputSize);
}
